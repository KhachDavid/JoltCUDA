#include "hip/hip_runtime.h"
#include "CudaMath.h"
#include "CollideSoftBodyVertices.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel using CUDA-friendly types
__global__ void CollideSoftBodyVerticesKernelCUDA(
    const float3 *d_inPositions,
    const float *d_inInvMass,
    float3 *d_outPositions,
    float3 *d_collisionPlane,
    float *d_largestPenetration,
    int *d_collidingShapeIndex,
    int numVertices,
    CudaMat44 inverse_transform,
    float3 half_extent,
    int collidingShapeIndex)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numVertices)
        return;

    float3 position = d_inPositions[i];
    float invMass = d_inInvMass[i];

    // For this example, simply copy the position
    d_outPositions[i] = position;

    if (invMass > 0.f)
    {
        // Transform to local space:
        float3 local_pos = MultiplyMat44Float3(inverse_transform, position);

        // Clamp local_pos:
        float3 neg_half_extent = make_float3(-half_extent.x, -half_extent.y, -half_extent.z);
        float3 temp = fminf3(local_pos, half_extent);
        float3 clamped_point = fmaxf3(temp, neg_half_extent);

        bool inside = (local_pos.x == clamped_point.x) &&
                      (local_pos.y == clamped_point.y) &&
                      (local_pos.z == clamped_point.z);

        if (inside)
        {
            // Compute penetration along each axis:
            float3 abs_local = fabsf3(local_pos);
            float3 delta;
            delta.x = half_extent.x - abs_local.x;
            delta.y = half_extent.y - abs_local.y;
            delta.z = half_extent.z - abs_local.z;

            int axis = 0;
            float penetration = delta.x;
            if (delta.y < penetration) { axis = 1; penetration = delta.y; }
            if (delta.z < penetration) { axis = 2; penetration = delta.z; }

            if (d_largestPenetration[i] < penetration)
            {
                d_largestPenetration[i] = penetration;
                float3 sign;
                sign.x = (local_pos.x < 0.f) ? -1.f : 1.f;
                sign.y = (local_pos.y < 0.f) ? -1.f : 1.f;
                sign.z = (local_pos.z < 0.f) ? -1.f : 1.f;

                float3 possible_normals[3] = {
                    make_float3(1.f, 0.f, 0.f),
                    make_float3(0.f, 1.f, 0.f),
                    make_float3(0.f, 0.f, 1.f)
                };

                float3 normal = make_float3(0.f, 0.f, 0.f);
                if (axis == 0)
                    normal = make_float3(sign.x * possible_normals[0].x, 0.f, 0.f);
                else if (axis == 1)
                    normal = make_float3(0.f, sign.y * possible_normals[1].y, 0.f);
                else if (axis == 2)
                    normal = make_float3(0.f, 0.f, sign.z * possible_normals[2].z);
                
                d_outPositions[i] = make_float3(half_extent.x * normal.x, half_extent.y * normal.y, half_extent.z * normal.z);
                d_collisionPlane[i] = normal;
                d_collidingShapeIndex[i] = collidingShapeIndex;
            }
        }
        else
        {
            float3 diff = local_pos - clamped_point;
            float norm_length = lengthF(diff);
            float penetration = -norm_length;
            if (d_largestPenetration[i] < penetration)
            {
                d_largestPenetration[i] = penetration;
                float3 norm = (norm_length > 0.f) ? diff / norm_length : make_float3(0.f,0.f,0.f);
                d_collisionPlane[i] = norm;
                d_collidingShapeIndex[i] = collidingShapeIndex;
                d_outPositions[i] = clamped_point;
            }
        }
    }
}

void LaunchCollideSoftBodyVerticesKernelCUDA(
    const float3* hPositions,
    const float* hInvMass,
    float3* hOutPositions,
    float3* hCollisionPlane,
    float* hLargestPenetration,
    int* h_collidingShapeIndex,
    int numVertices,
    const float hMat[16],
    const float hHalfExtent[3],
    int collidingShapeIndex)
{
    printf("Launching CollideSoftBodyVerticesKernelCUDA...\n");
    float3 *d_inPositions = nullptr, *d_outPositions = nullptr, *d_collisionPlane = nullptr;
    float  *d_inInvMass = nullptr, *d_largestPenetration = nullptr;
    int    *d_collidingShapeIndex = nullptr;

    size_t posSize = numVertices * sizeof(float3);
    size_t floatSize = numVertices * sizeof(float);
    size_t intSize = numVertices * sizeof(int);

    hipError_t err = hipMalloc(&d_inPositions, posSize);
    if (err != hipSuccess) { printf("hipMalloc d_inPositions: %s\n", hipGetErrorString(err)); return; }
    err = hipMalloc(&d_inInvMass, floatSize);
    if (err != hipSuccess) { printf("hipMalloc d_inInvMass: %s\n", hipGetErrorString(err)); return; }
    err = hipMalloc(&d_outPositions, posSize);
    if (err != hipSuccess) { printf("hipMalloc d_outPositions: %s\n", hipGetErrorString(err)); return; }
    err = hipMalloc(&d_collisionPlane, posSize);
    if (err != hipSuccess) { printf("hipMalloc d_collisionPlane: %s\n", hipGetErrorString(err)); return; }
    err = hipMalloc(&d_largestPenetration, floatSize);
    if (err != hipSuccess) { printf("hipMalloc d_largestPenetration: %s\n", hipGetErrorString(err)); return; }
    err = hipMalloc(&d_collidingShapeIndex, intSize);
    if (err != hipSuccess) { printf("hipMalloc d_collidingShapeIndex: %s\n", hipGetErrorString(err)); return; }

    err = hipMemcpy(d_inPositions, hPositions, posSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) { printf("hipMemcpy d_inPositions: %s\n", hipGetErrorString(err)); return; }
    err = hipMemcpy(d_inInvMass, hInvMass, floatSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) { printf("hipMemcpy d_inInvMass: %s\n", hipGetErrorString(err)); return; }
    err = hipMemcpy(d_largestPenetration, hLargestPenetration, floatSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) { printf("hipMemcpy d_largestPenetration: %s\n", hipGetErrorString(err)); return; }
    err = hipMemcpy(d_collidingShapeIndex, h_collidingShapeIndex, intSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) { printf("hipMemcpy d_collidingShapeIndex: %s\n", hipGetErrorString(err)); return; }

    CudaMat44 d_inverse;
    for (int i = 0; i < 16; ++i)
        d_inverse.m[i] = hMat[i];

    float3 d_halfExtent = make_float3(hHalfExtent[0], hHalfExtent[1], hHalfExtent[2]);

    int blockSize = 256;
    int numBlocks = (numVertices + blockSize - 1) / blockSize;
    CollideSoftBodyVerticesKernelCUDA<<<numBlocks, blockSize>>>(
         d_inPositions, d_inInvMass, d_outPositions, d_collisionPlane,
         d_largestPenetration, d_collidingShapeIndex,
         numVertices, d_inverse, d_halfExtent, collidingShapeIndex);
    err = hipGetLastError();
    if (err != hipSuccess) 
    {
         printf("Kernel launch error: %s\n", hipGetErrorString(err));
         return;
    }
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess)
    {
         printf("Kernel after sync error: %s\n", hipGetErrorString(err));
         return;
    }

    err = hipMemcpy((void*)hOutPositions, d_outPositions, posSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) { printf("hipMemcpy hOutPositions: %s\n", hipGetErrorString(err)); return; }
    err = hipMemcpy((void*)hCollisionPlane, d_collisionPlane, posSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) { printf("hipMemcpy hCollisionPlane: %s\n", hipGetErrorString(err)); return; }
    err = hipMemcpy((void*)hLargestPenetration, d_largestPenetration, floatSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) { printf("hipMemcpy hLargestPenetration: %s\n", hipGetErrorString(err)); return; }
    err = hipMemcpy((void*)h_collidingShapeIndex, d_collidingShapeIndex, intSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) { printf("hipMemcpy d_collidingShapeIndex: %s\n", hipGetErrorString(err)); return; }

    hipFree(d_inPositions);
    hipFree(d_inInvMass);
    hipFree(d_outPositions);
    hipFree(d_collisionPlane);
    hipFree(d_largestPenetration);
    hipFree(d_collidingShapeIndex);
}