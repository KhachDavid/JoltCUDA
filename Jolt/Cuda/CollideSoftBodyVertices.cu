#include "hip/hip_runtime.h"
__global__ void CollideSoftBodyVerticesKernelCUDA(
    const float3 *__restrict__ d_inPositions,
    const float *__restrict__ d_inInvMass,
    float3 *__restrict__ d_outPositions,
    float3 *__restrict__ d_collisionPlane,
    float *__restrict__ d_largestPenetration,
    int *__restrict__ d_collidingShapeIndex,
    int numVertices,
    CudaMat44 inverse_transform,
    float3 half_extent,
    int collidingShapeIndex)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numVertices) return;

    float3 position = d_inPositions[i];
    float invMass = d_inInvMass[i];

    // Simply copy position by default
    d_outPositions[i] = position;

    if (invMass > 0.f)
    {
        // Transform to local space:
        float3 local_pos = MultiplyMat44Float3(inverse_transform, position);

        // Clamp local_pos:
        float3 neg_half_extent = -half_extent;
        float3 clamped_point = fminf3(fmaxf3(local_pos, neg_half_extent), half_extent);

        bool inside = (local_pos.x == clamped_point.x) &&
                      (local_pos.y == clamped_point.y) &&
                      (local_pos.z == clamped_point.z);

        if (inside)
        {
            float3 delta = make_float3(half_extent.x - fabsf(local_pos.x),
                                       half_extent.y - fabsf(local_pos.y),
                                       half_extent.z - fabsf(local_pos.z));

            // Find the axis with the smallest penetration
            int axis = (delta.y < delta.x) ? ((delta.z < delta.y) ? 2 : 1) : ((delta.z < delta.x) ? 2 : 0);
            float penetration = delta[axis];

            if (d_largestPenetration[i] < penetration)
            {
                d_largestPenetration[i] = penetration;
                
                float sign = (local_pos[axis] < 0.f) ? -1.f : 1.f;
                float3 normal = make_float3(0.f, 0.f, 0.f);
                normal[axis] = sign;

                d_outPositions[i] = normal * half_extent;
                d_collisionPlane[i] = normal;
                d_collidingShapeIndex[i] = collidingShapeIndex;
            }
        }
        else
        {
            float3 diff = local_pos - clamped_point;
            float norm_length = lengthF(diff);
            float penetration = -norm_length;
            
            if (d_largestPenetration[i] < penetration)
            {
                d_largestPenetration[i] = penetration;
                d_collisionPlane[i] = (norm_length > 0.f) ? diff / norm_length : make_float3(0.f, 0.f, 0.f);
                d_collidingShapeIndex[i] = collidingShapeIndex;
                d_outPositions[i] = clamped_point;
            }
        }
    }
}

// Optimized Kernel Launch Function
void LaunchCollideSoftBodyVerticesKernelCUDA(
    const float3 *hPositions,
    const float *hInvMass,
    float3 *hOutPositions,
    float3 *hCollisionPlane,
    float *hLargestPenetration,
    int *h_collidingShapeIndex,
    int numVertices,
    const float hMat[16],
    const float hHalfExtent[3],
    int collidingShapeIndex)
{
    float3 *d_inPositions = nullptr, *d_outPositions = nullptr, *d_collisionPlane = nullptr;
    float *d_inInvMass = nullptr, *d_largestPenetration = nullptr;
    int *d_collidingShapeIndex = nullptr;

    size_t posSize = numVertices * sizeof(float3);
    size_t floatSize = numVertices * sizeof(float);
    size_t intSize = numVertices * sizeof(int);

    hipMalloc(&d_inPositions, posSize);
    hipMalloc(&d_inInvMass, floatSize);
    hipMalloc(&d_outPositions, posSize);
    hipMalloc(&d_collisionPlane, posSize);
    hipMalloc(&d_largestPenetration, floatSize);
    hipMalloc(&d_collidingShapeIndex, intSize);

    hipMemcpy(d_inPositions, hPositions, posSize, hipMemcpyHostToDevice);
    hipMemcpy(d_inInvMass, hInvMass, floatSize, hipMemcpyHostToDevice);
    hipMemcpy(d_largestPenetration, hLargestPenetration, floatSize, hipMemcpyHostToDevice);
    hipMemcpy(d_collidingShapeIndex, h_collidingShapeIndex, intSize, hipMemcpyHostToDevice);

    CudaMat44 d_inverse;
    for (int i = 0; i < 16; ++i)
        d_inverse.m[i] = hMat[i];

    float3 d_halfExtent = make_float3(hHalfExtent[0], hHalfExtent[1], hHalfExtent[2]);

    // **Optimized Block Size**
    int blockSize = 128;  // Tuned for best occupancy
    int numBlocks = (numVertices + blockSize - 1) / blockSize;

    CollideSoftBodyVerticesKernelCUDA<<<numBlocks, blockSize>>>(
        d_inPositions, d_inInvMass, d_outPositions, d_collisionPlane,
        d_largestPenetration, d_collidingShapeIndex, numVertices, d_inverse, d_halfExtent, collidingShapeIndex);
    
    hipDeviceSynchronize();

    hipMemcpy(hOutPositions, d_outPositions, posSize, hipMemcpyDeviceToHost);
    hipMemcpy(hCollisionPlane, d_collisionPlane, posSize, hipMemcpyDeviceToHost);
    hipMemcpy(hLargestPenetration, d_largestPenetration, floatSize, hipMemcpyDeviceToHost);
    hipMemcpy(h_collidingShapeIndex, d_collidingShapeIndex, intSize, hipMemcpyDeviceToHost);

    hipFree(d_inPositions);
    hipFree(d_inInvMass);
    hipFree(d_outPositions);
    hipFree(d_collisionPlane);
    hipFree(d_largestPenetration);
    hipFree(d_collidingShapeIndex);
}