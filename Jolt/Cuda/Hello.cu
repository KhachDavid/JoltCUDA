#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA Kernel to print "Hello, World!" on GPU
__global__ void HelloWorldKernel() {
    printf("Hello, hi from CUDA! Thread ID: %d\n", threadIdx.x);
}

// Function to launch the kernel
extern "C" void LaunchHelloKernel() {
    HelloWorldKernel<<<1, 10>>>();  // Launch with 10 threads
    hipDeviceSynchronize();        // Ensure the kernel completes execution
}
