#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA Kernel to print "Hello, World!" on GPU
__global__ void HelloWorldKernel() {
    printf("Hello, World from CUDA! Thread ID: %d\n", threadIdx.x);
}
