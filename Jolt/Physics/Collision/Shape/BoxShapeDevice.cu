
#include <hip/hip_runtime.h>


// void BoxShape::CollideSoftBodyVertices(Mat44Arg inCenterOfMassTransform, Vec3Arg inScale, const CollideSoftBodyVertexIterator &inVertices, uint inNumVertices, int inCollidingShapeIndex) const
// {
// 	// Print the inCenterOfMassTransform and what 
// 	printf("Counter: %d\n", counter);
// 	for (int i = 0; i < 4; i++)
// 	{
// 		// print the first row
// 		printf("%f ", inCenterOfMassTransform.);
// 	}
// 	counter++;
// 	Mat44 inverse_transform = inCenterOfMassTransform.InversedRotationTranslation();
// 	Vec3 half_extent = inScale.Abs() * mHalfExtent;

// 	for (CollideSoftBodyVertexIterator v = inVertices, sbv_end = inVertices + inNumVertices; v != sbv_end; ++v)
// 		if (v.GetInvMass() > 0.0f)
// 		{
// 			// Convert to local space
// 			Vec3 local_pos = inverse_transform * v.GetPosition();

// 			// Clamp point to inside box
// 			Vec3 clamped_point = Vec3::sMax(Vec3::sMin(local_pos, half_extent), -half_extent);

// 			// Test if point was inside
// 			if (clamped_point == local_pos)
// 			{
// 				// Calculate closest distance to surface
// 				Vec3 delta = half_extent - local_pos.Abs();
// 				int index = delta.GetLowestComponentIndex();
// 				float penetration = delta[index];
// 				if (v.UpdatePenetration(penetration))
// 				{
// 					// Calculate contact point and normal
// 					Vec3 possible_normals[] = { Vec3::sAxisX(), Vec3::sAxisY(), Vec3::sAxisZ() };
// 					Vec3 normal = local_pos.GetSign() * possible_normals[index];
// 					Vec3 point = normal * half_extent;

// 					// Store collision
// 					v.SetCollision(Plane::sFromPointAndNormal(point, normal).GetTransformed(inCenterOfMassTransform), inCollidingShapeIndex);
// 				}
// 			}
// 			else
// 			{
// 				// Calculate normal
// 				Vec3 normal = local_pos - clamped_point;
// 				float normal_length = normal.Length();

// 				// Penetration will be negative since we're not penetrating
// 				float penetration = -normal_length;
// 				if (v.UpdatePenetration(penetration))
// 				{
// 					normal /= normal_length;

// 					// Store collision
// 					v.SetCollision(Plane::sFromPointAndNormal(clamped_point, normal).GetTransformed(inCenterOfMassTransform), inCollidingShapeIndex);
// 				}
// 			}
// 		}
// }

// write a cude hello world program and add all the necessary files to the project

